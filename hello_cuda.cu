
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello()
{
    printf("Hello World from GPU!\n");
}

int main()
{
    cuda_hello<<<1, 1>>>();
    hipDeviceSynchronize(); // Wait for the GPU launched work to complete
    return 0;
}