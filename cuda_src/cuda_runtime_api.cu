#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>

int main(int argc, char const *argv[])
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Device count: %d\n", deviceCount);
    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        printf("Device %d: %s\n", i, deviceProp.name);
        printf("  Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("  Multiprocessor count: %d\n", deviceProp.multiProcessorCount);
        printf("  Memory bus width: %d bits\n", deviceProp.memoryBusWidth);
        printf("  Peak memory bandwidth: %f GB/s\n", 2.0 * deviceProp.memoryClockRate * deviceProp.memoryBusWidth / 8 / 1e6);
        printf("  Total global memory: %lu bytes\n", deviceProp.totalGlobalMem);
        printf("  Shared memory per block: %lu bytes\n", deviceProp.sharedMemPerBlock);
        printf("  Warp size: %d\n", deviceProp.warpSize);
        // max active blocks per SM, which are the limit of blocks that can be executed simultaneously on an SM
        printf("  Max blocks per multiprocessor: %d\n", deviceProp.maxBlocksPerMultiProcessor);
        printf("  Max warps per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor / deviceProp.warpSize);
        printf("  Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
        printf("  Max threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
        printf("  Max registers per block: %d\n", deviceProp.regsPerBlock);
        printf("  Max threads dimensions: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf("  Max grid size: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        printf("  Clock rate: %d kHz\n", deviceProp.clockRate);
        printf("  Total constant memory: %lu bytes\n", deviceProp.totalConstMem);
        // max block per SM
    }
    return 0;
}
